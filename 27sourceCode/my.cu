#include "hip/hip_runtime.h"
#include "YUV.SJTU_headers.h"
//#include "upscale.h"

#define  HF				(8.0f)
#define  a				(-0.50000000f)
__inline__ __device__ float cubic(float x){
	float abs_x = fabsf(x);

	if (abs_x <= 1.0f){
		return (a + 2)*abs_x *abs_x *abs_x - (a + 3)*abs_x *abs_x + 1;
	}
	else if (abs_x < 2.0f){
		return a * abs_x *abs_x *abs_x - 5 * a *abs_x *abs_x + 8 * a *abs_x - 4 * a;
	}
	else {
		return 0.0f;
	}
}
__inline__ __device__ float linear(float x){
	return x;
}


__global__ void DownSample(unsigned char * In, int iw, int ih,
	float * downbicu, int dw, int dh,
	float downscale_x, float downscale_y
	) {
	const int dstBaseX = blockIdx.x * blockDim.x + threadIdx.x;
	const int dstBaseY = blockIdx.y * blockDim.y + threadIdx.y;

	float downbicu_value = 0.0f;
	//float downbili_value = 0.0f;

	float srcY, srcX;
	int srcY0, srcX0; int srcY1, srcX1; int srcY2, srcX2; int srcY3, srcX3;


	float cubic_X[4], cubic_Y[4];
	//float linear_X[2], linear_Y[2];

	srcX = (dstBaseX + 1) / downscale_x + 0.5f*(1.0f - 1.0f / downscale_x) - 1;
	srcY = (dstBaseY + 1) / downscale_y + 0.5f*(1.0f - 1.0f / downscale_y) - 1;

	srcX0 = floorf(srcX) - 1;
	srcY0 = floorf(srcY) - 1;
	srcX1 = srcX0 + 1;
	srcY1 = srcY0 + 1;
	srcX2 = srcX0 + 2;
	srcY2 = srcY0 + 2;
	srcX3 = srcX0 + 3;
	srcY3 = srcY0 + 3;

	cubic_X[0] = cubic(srcX - srcX0);
	cubic_X[1] = cubic(srcX - srcX1);
	cubic_X[2] = cubic(srcX - srcX2);
	cubic_X[3] = cubic(srcX - srcX3);

	cubic_Y[0] = cubic(srcY - srcY0);
	cubic_Y[1] = cubic(srcY - srcY1);
	cubic_Y[2] = cubic(srcY - srcY2);
	cubic_Y[3] = cubic(srcY - srcY3);

	//linear_X[0] = linear(srcX2 - srcX);
	//linear_X[1] = linear(srcX - srcX1);

	//linear_Y[0] = linear(srcY2 - srcY);
	//linear_Y[1] = linear(srcY - srcY1);


	srcX0 = min(max(srcX0, 0), iw - 1);// (srcX0 < 0) ? 0 : srcX0;
	srcY0 = min(max(srcY0, 0), ih - 1);// (srcY0 < 0) ? 0 : srcY0;
	srcX1 = min(max(srcX1, 0), iw - 1);// (srcX1 < 0) ? 0 : srcX1;
	srcY1 = min(max(srcY1, 0), ih - 1);//(srcY1 < 0) ? 0 : srcY1;
	srcX2 = min(max(srcX2, 0), iw - 1);// (srcX2 > validW - 1) ? validW - 1 : srcX2;
	srcY2 = min(max(srcY2, 0), ih - 1);//(srcY2 > validH - 1) ? validH - 1 : srcY2;
	srcX3 = min(max(srcX3, 0), iw - 1);// (srcX3 > validW - 1) ? validW - 1 : srcX3;
	srcY3 = min(max(srcY3, 0), ih - 1);//(srcY3 > validH - 1) ? validH - 1 : srcY3;

	downbicu_value =
		cubic_X[0] * cubic_Y[0] * In[srcY0 * iw + srcX0] +
		cubic_X[0] * cubic_Y[1] * In[srcY1 * iw + srcX0] +
		cubic_X[0] * cubic_Y[2] * In[srcY2 * iw + srcX0] +
		cubic_X[0] * cubic_Y[3] * In[srcY3 * iw + srcX0] +
		cubic_X[1] * cubic_Y[0] * In[srcY0 * iw + srcX1] +
		cubic_X[1] * cubic_Y[1] * In[srcY1 * iw + srcX1] +
		cubic_X[1] * cubic_Y[2] * In[srcY2 * iw + srcX1] +
		cubic_X[1] * cubic_Y[3] * In[srcY3 * iw + srcX1] +
		cubic_X[2] * cubic_Y[0] * In[srcY0 * iw + srcX2] +
		cubic_X[2] * cubic_Y[1] * In[srcY1 * iw + srcX2] +
		cubic_X[2] * cubic_Y[2] * In[srcY2 * iw + srcX2] +
		cubic_X[2] * cubic_Y[3] * In[srcY3 * iw + srcX2] +
		cubic_X[3] * cubic_Y[0] * In[srcY0 * iw + srcX3] +
		cubic_X[3] * cubic_Y[1] * In[srcY1 * iw + srcX3] +
		cubic_X[3] * cubic_Y[2] * In[srcY2 * iw + srcX3] +
		cubic_X[3] * cubic_Y[3] * In[srcY3 * iw + srcX3];

	//downbili_value =
	//	linear_X[0] * linear_Y[0] * In[srcY1 * iw + srcX1] +
	//	linear_X[0] * linear_Y[1] * In[srcY2 * iw + srcX1] +
	//	linear_X[1] * linear_Y[0] * In[srcY1 * iw + srcX2] +
	//	linear_X[1] * linear_Y[1] * In[srcY2 * iw + srcX2];

	if (dstBaseY < dh && dstBaseX < dw){
		downbicu[dstBaseY * dw + dstBaseX] = downbicu_value;
		//downbili[dstBaseY * dw + dstBaseX] = downbili_value;
	}
}

__global__ void UpEnhanceLL(
	float * downbicu, int dw, int dh,
	unsigned char *In, unsigned char * Out, int ow, int oh,
	float upscale_x, float upscale_y
	){
	const int dstBaseX = blockIdx.x * blockDim.x + threadIdx.x;
	const int dstBaseY = blockIdx.y * blockDim.y + threadIdx.y;

	float upbicu_value = 0.0f;
	//float upbili_value = 0.0f;


	float srcY, srcX;

	int srcY0, srcX0; int srcY1, srcX1; int srcY2, srcX2; int srcY3, srcX3;

	float cubic_X[4], cubic_Y[4];
	//float linear_X[2], linear_Y[2];

	if (dstBaseX < ow / 2 - 1){
	// i just use the local coordinate instead of global coordinate....this may cause some block effect ....let's see
	srcX = (dstBaseX + 1) / upscale_x + 0.5f*(1.0f - 1.0f / upscale_x) - 1;
	srcY = (dstBaseY + 1) / upscale_y + 0.5f*(1.0f - 1.0f / upscale_y) - 1;

	srcX0 = floorf(srcX) - 1;
	srcY0 = floorf(srcY) - 1;
	srcX1 = srcX0 + 1;
	srcY1 = srcY0 + 1;
	srcX2 = srcX0 + 2;
	srcY2 = srcY0 + 2;
	srcX3 = srcX0 + 3;
	srcY3 = srcY0 + 3;

	cubic_X[0] = cubic(srcX - srcX0);
	cubic_X[1] = cubic(srcX - srcX1);
	cubic_X[2] = cubic(srcX - srcX2);
	cubic_X[3] = cubic(srcX - srcX3);

	cubic_Y[0] = cubic(srcY - srcY0);
	cubic_Y[1] = cubic(srcY - srcY1);
	cubic_Y[2] = cubic(srcY - srcY2);
	cubic_Y[3] = cubic(srcY - srcY3);

	//linear_X[0] = linear(srcX2 - srcX);
	//linear_X[1] = linear(srcX - srcX1);

	//linear_Y[0] = linear(srcY2 - srcY);
	//linear_Y[1] = linear(srcY - srcY1);

	srcX0 = min(max(srcX0, 0), dw - 1);// (srcX0 < 0) ? 0 : srcX0;
	srcY0 = min(max(srcY0, 0), dh - 1);// (srcY0 < 0) ? 0 : srcY0;
	srcX1 = min(max(srcX1, 0), dw - 1);// (srcX1 < 0) ? 0 : srcX1;
	srcY1 = min(max(srcY1, 0), dh - 1);//(srcY1 < 0) ? 0 : srcY1;
	srcX2 = min(max(srcX2, 0), dw - 1);// (srcX2 > validW - 1) ? validW - 1 : srcX2;
	srcY2 = min(max(srcY2, 0), dh - 1);//(srcY2 > validH - 1) ? validH - 1 : srcY2;
	srcX3 = min(max(srcX3, 0), dw - 1);// (srcX3 > validW - 1) ? validW - 1 : srcX3;
	srcY3 = min(max(srcY3, 0), dh - 1);//(srcY3 > validH - 1) ? validH - 1 : srcY3;


	upbicu_value =
		cubic_X[0] * cubic_Y[0] * downbicu[srcY0 * dw + srcX0] +
		cubic_X[0] * cubic_Y[1] * downbicu[srcY1 * dw + srcX0] +
		cubic_X[0] * cubic_Y[2] * downbicu[srcY2 * dw + srcX0] +
		cubic_X[0] * cubic_Y[3] * downbicu[srcY3 * dw + srcX0] +
		cubic_X[1] * cubic_Y[0] * downbicu[srcY0 * dw + srcX1] +
		cubic_X[1] * cubic_Y[1] * downbicu[srcY1 * dw + srcX1] +
		cubic_X[1] * cubic_Y[2] * downbicu[srcY2 * dw + srcX1] +
		cubic_X[1] * cubic_Y[3] * downbicu[srcY3 * dw + srcX1] +
		cubic_X[2] * cubic_Y[0] * downbicu[srcY0 * dw + srcX2] +
		cubic_X[2] * cubic_Y[1] * downbicu[srcY1 * dw + srcX2] +
		cubic_X[2] * cubic_Y[2] * downbicu[srcY2 * dw + srcX2] +
		cubic_X[2] * cubic_Y[3] * downbicu[srcY3 * dw + srcX2] +
		cubic_X[3] * cubic_Y[0] * downbicu[srcY0 * dw + srcX3] +
		cubic_X[3] * cubic_Y[1] * downbicu[srcY1 * dw + srcX3] +
		cubic_X[3] * cubic_Y[2] * downbicu[srcY2 * dw + srcX3] +
		cubic_X[3] * cubic_Y[3] * downbicu[srcY3 * dw + srcX3];

	//upbili_value =
	//	linear_X[0] * linear_Y[0] * downbili[srcY1 * dw + srcX1] +
	//	linear_X[0] * linear_Y[1] * downbili[srcY2 * dw + srcX1] +
	//	linear_X[1] * linear_Y[0] * downbili[srcY1 * dw + srcX2] +
	//	linear_X[1] * linear_Y[1] * downbili[srcY2 * dw + srcX2];


	upbicu_value = In[dstBaseY * ow + dstBaseX] + HF*(In[dstBaseY * ow + dstBaseX] - upbicu_value);
	}
	else if (dstBaseX == ow / 2 - 1 || dstBaseX == ow / 2){
		upbicu_value = 255;
	}
	else{
		upbicu_value = In[dstBaseY * ow + dstBaseX - ow / 2];
	}

	if (dstBaseY < oh && dstBaseX < ow){
		Out[dstBaseY * ow + dstBaseX] = min(max(unsigned int(upbicu_value), 0), 255);
	}
}

void OptBicu_BicuexeckernelLL(unsigned char * In, int iw, int ih,
	float * downbicu, int dw, int dh, float downscale_x, float downscale_y,
	unsigned char * Out, int ow, int oh, float upscale_x, float upscale_y
	){

	static dim3 grid;
	static dim3 block;

	block.x = 32;
	block.y = 32;
	block.z = 1;
	grid.x = (dw + block.x - 1) / block.x;
	grid.y = (dh + block.y - 1) / block.y;
	grid.z = 1;

	DownSample << <grid, block >> >(In, iw, ih, downbicu, dw, dh, downscale_x, downscale_y);
	
	grid.x = (ow + block.x - 1) / block.x;
	grid.y = (oh + block.y - 1) / block.y;
	grid.z = 1;


	//Sheme Four:
		//left side ==>  I + 4 *(I - upbicu(downbicu(I)))
		//right side ==> I
	UpEnhanceLL << <grid, block >> >(downbicu, dw, dh,In, Out, ow, oh, upscale_x, upscale_y);


}
 


__global__ void UpEnhanceLR(
	float * downbicu, int dw, int dh,
	unsigned char *In, unsigned char * Out, int ow, int oh,
	float upscale_x, float upscale_y
	){
	const int dstBaseX = blockIdx.x * blockDim.x + threadIdx.x;
	const int dstBaseY = blockIdx.y * blockDim.y + threadIdx.y;

	float upbicu_value = 0.0f;
	//float upbili_value = 0.0f;


	float srcY, srcX;

	int srcY0, srcX0; int srcY1, srcX1; int srcY2, srcX2; int srcY3, srcX3;

	float cubic_X[4], cubic_Y[4];
	//float linear_X[2], linear_Y[2];


	// i just use the local coordinate instead of global coordinate....this may cause some block effect ....let's see
	srcX = (dstBaseX + 1) / upscale_x + 0.5f*(1.0f - 1.0f / upscale_x) - 1;
	srcY = (dstBaseY + 1) / upscale_y + 0.5f*(1.0f - 1.0f / upscale_y) - 1;

	srcX0 = floorf(srcX) - 1;
	srcY0 = floorf(srcY) - 1;
	srcX1 = srcX0 + 1;
	srcY1 = srcY0 + 1;
	srcX2 = srcX0 + 2;
	srcY2 = srcY0 + 2;
	srcX3 = srcX0 + 3;
	srcY3 = srcY0 + 3;

	cubic_X[0] = cubic(srcX - srcX0);
	cubic_X[1] = cubic(srcX - srcX1);
	cubic_X[2] = cubic(srcX - srcX2);
	cubic_X[3] = cubic(srcX - srcX3);

	cubic_Y[0] = cubic(srcY - srcY0);
	cubic_Y[1] = cubic(srcY - srcY1);
	cubic_Y[2] = cubic(srcY - srcY2);
	cubic_Y[3] = cubic(srcY - srcY3);

	//linear_X[0] = linear(srcX2 - srcX);
	//linear_X[1] = linear(srcX - srcX1);

	//linear_Y[0] = linear(srcY2 - srcY);
	//linear_Y[1] = linear(srcY - srcY1);

	srcX0 = min(max(srcX0, 0), dw - 1);// (srcX0 < 0) ? 0 : srcX0;
	srcY0 = min(max(srcY0, 0), dh - 1);// (srcY0 < 0) ? 0 : srcY0;
	srcX1 = min(max(srcX1, 0), dw - 1);// (srcX1 < 0) ? 0 : srcX1;
	srcY1 = min(max(srcY1, 0), dh - 1);//(srcY1 < 0) ? 0 : srcY1;
	srcX2 = min(max(srcX2, 0), dw - 1);// (srcX2 > validW - 1) ? validW - 1 : srcX2;
	srcY2 = min(max(srcY2, 0), dh - 1);//(srcY2 > validH - 1) ? validH - 1 : srcY2;
	srcX3 = min(max(srcX3, 0), dw - 1);// (srcX3 > validW - 1) ? validW - 1 : srcX3;
	srcY3 = min(max(srcY3, 0), dh - 1);//(srcY3 > validH - 1) ? validH - 1 : srcY3;


	upbicu_value =
		cubic_X[0] * cubic_Y[0] * downbicu[srcY0 * dw + srcX0] +
		cubic_X[0] * cubic_Y[1] * downbicu[srcY1 * dw + srcX0] +
		cubic_X[0] * cubic_Y[2] * downbicu[srcY2 * dw + srcX0] +
		cubic_X[0] * cubic_Y[3] * downbicu[srcY3 * dw + srcX0] +
		cubic_X[1] * cubic_Y[0] * downbicu[srcY0 * dw + srcX1] +
		cubic_X[1] * cubic_Y[1] * downbicu[srcY1 * dw + srcX1] +
		cubic_X[1] * cubic_Y[2] * downbicu[srcY2 * dw + srcX1] +
		cubic_X[1] * cubic_Y[3] * downbicu[srcY3 * dw + srcX1] +
		cubic_X[2] * cubic_Y[0] * downbicu[srcY0 * dw + srcX2] +
		cubic_X[2] * cubic_Y[1] * downbicu[srcY1 * dw + srcX2] +
		cubic_X[2] * cubic_Y[2] * downbicu[srcY2 * dw + srcX2] +
		cubic_X[2] * cubic_Y[3] * downbicu[srcY3 * dw + srcX2] +
		cubic_X[3] * cubic_Y[0] * downbicu[srcY0 * dw + srcX3] +
		cubic_X[3] * cubic_Y[1] * downbicu[srcY1 * dw + srcX3] +
		cubic_X[3] * cubic_Y[2] * downbicu[srcY2 * dw + srcX3] +
		cubic_X[3] * cubic_Y[3] * downbicu[srcY3 * dw + srcX3];

	//upbili_value =
	//	linear_X[0] * linear_Y[0] * downbili[srcY1 * dw + srcX1] +
	//	linear_X[0] * linear_Y[1] * downbili[srcY2 * dw + srcX1] +
	//	linear_X[1] * linear_Y[0] * downbili[srcY1 * dw + srcX2] +
	//	linear_X[1] * linear_Y[1] * downbili[srcY2 * dw + srcX2];

	if (dstBaseX < ow / 2 - 1){
		upbicu_value = In[dstBaseY * ow + dstBaseX] + HF*(In[dstBaseY * ow + dstBaseX] - upbicu_value);
	}
	else if (dstBaseX == ow / 2 - 1 || dstBaseX == ow / 2){
		upbicu_value = 255;
	}
	else{
		upbicu_value = In[dstBaseY * ow + dstBaseX];
	}

	if (dstBaseY < oh && dstBaseX < ow){
		Out[dstBaseY * ow + dstBaseX] = min(max(unsigned int(upbicu_value), 0), 255);
	}
}


void OptBicu_BicuexeckernelLR(unsigned char * In, int iw, int ih,
	float * downbicu, int dw, int dh, float downscale_x, float downscale_y,
	unsigned char * Out, int ow, int oh, float upscale_x, float upscale_y
	){

	static dim3 grid;
	static dim3 block;

	block.x = 32;
	block.y = 32;
	block.z = 1;
	grid.x = (dw + block.x - 1) / block.x;
	grid.y = (dh + block.y - 1) / block.y;
	grid.z = 1;

	DownSample << <grid, block >> >(In, iw, ih, downbicu, dw, dh, downscale_x, downscale_y);

	grid.x = (ow + block.x - 1) / block.x;
	grid.y = (oh + block.y - 1) / block.y;
	grid.z = 1;


	//Sheme Four:
	//left side ==>  I + 4 *(I - upbicu(downbicu(I)))
	//right side ==> I
	UpEnhanceLR << <grid, block >> >(downbicu, dw, dh, In, Out, ow, oh, upscale_x, upscale_y);


}

__global__ void UpEnhance(
	float * downbicu, int dw, int dh,
	unsigned char *In, unsigned char * Out, int ow, int oh,
	float upscale_x, float upscale_y
	){
	const int dstBaseX = blockIdx.x * blockDim.x + threadIdx.x;
	const int dstBaseY = blockIdx.y * blockDim.y + threadIdx.y;

	float upbicu_value = 0.0f;
	//float upbili_value = 0.0f;


	float srcY, srcX;

	int srcY0, srcX0; int srcY1, srcX1; int srcY2, srcX2; int srcY3, srcX3;

	float cubic_X[4], cubic_Y[4];
	//float linear_X[2], linear_Y[2];


	// i just use the local coordinate instead of global coordinate....this may cause some block effect ....let's see
	srcX = (dstBaseX + 1) / upscale_x + 0.5f*(1.0f - 1.0f / upscale_x) - 1;
	srcY = (dstBaseY + 1) / upscale_y + 0.5f*(1.0f - 1.0f / upscale_y) - 1;

	srcX0 = floorf(srcX) - 1;
	srcY0 = floorf(srcY) - 1;
	srcX1 = srcX0 + 1;
	srcY1 = srcY0 + 1;
	srcX2 = srcX0 + 2;
	srcY2 = srcY0 + 2;
	srcX3 = srcX0 + 3;
	srcY3 = srcY0 + 3;

	cubic_X[0] = cubic(srcX - srcX0);
	cubic_X[1] = cubic(srcX - srcX1);
	cubic_X[2] = cubic(srcX - srcX2);
	cubic_X[3] = cubic(srcX - srcX3);

	cubic_Y[0] = cubic(srcY - srcY0);
	cubic_Y[1] = cubic(srcY - srcY1);
	cubic_Y[2] = cubic(srcY - srcY2);
	cubic_Y[3] = cubic(srcY - srcY3);

	//linear_X[0] = linear(srcX2 - srcX);
	//linear_X[1] = linear(srcX - srcX1);

	//linear_Y[0] = linear(srcY2 - srcY);
	//linear_Y[1] = linear(srcY - srcY1);

	srcX0 = min(max(srcX0, 0), dw - 1);// (srcX0 < 0) ? 0 : srcX0;
	srcY0 = min(max(srcY0, 0), dh - 1);// (srcY0 < 0) ? 0 : srcY0;
	srcX1 = min(max(srcX1, 0), dw - 1);// (srcX1 < 0) ? 0 : srcX1;
	srcY1 = min(max(srcY1, 0), dh - 1);//(srcY1 < 0) ? 0 : srcY1;
	srcX2 = min(max(srcX2, 0), dw - 1);// (srcX2 > validW - 1) ? validW - 1 : srcX2;
	srcY2 = min(max(srcY2, 0), dh - 1);//(srcY2 > validH - 1) ? validH - 1 : srcY2;
	srcX3 = min(max(srcX3, 0), dw - 1);// (srcX3 > validW - 1) ? validW - 1 : srcX3;
	srcY3 = min(max(srcY3, 0), dh - 1);//(srcY3 > validH - 1) ? validH - 1 : srcY3;


	upbicu_value =
		cubic_X[0] * cubic_Y[0] * downbicu[srcY0 * dw + srcX0] +
		cubic_X[0] * cubic_Y[1] * downbicu[srcY1 * dw + srcX0] +
		cubic_X[0] * cubic_Y[2] * downbicu[srcY2 * dw + srcX0] +
		cubic_X[0] * cubic_Y[3] * downbicu[srcY3 * dw + srcX0] +
		cubic_X[1] * cubic_Y[0] * downbicu[srcY0 * dw + srcX1] +
		cubic_X[1] * cubic_Y[1] * downbicu[srcY1 * dw + srcX1] +
		cubic_X[1] * cubic_Y[2] * downbicu[srcY2 * dw + srcX1] +
		cubic_X[1] * cubic_Y[3] * downbicu[srcY3 * dw + srcX1] +
		cubic_X[2] * cubic_Y[0] * downbicu[srcY0 * dw + srcX2] +
		cubic_X[2] * cubic_Y[1] * downbicu[srcY1 * dw + srcX2] +
		cubic_X[2] * cubic_Y[2] * downbicu[srcY2 * dw + srcX2] +
		cubic_X[2] * cubic_Y[3] * downbicu[srcY3 * dw + srcX2] +
		cubic_X[3] * cubic_Y[0] * downbicu[srcY0 * dw + srcX3] +
		cubic_X[3] * cubic_Y[1] * downbicu[srcY1 * dw + srcX3] +
		cubic_X[3] * cubic_Y[2] * downbicu[srcY2 * dw + srcX3] +
		cubic_X[3] * cubic_Y[3] * downbicu[srcY3 * dw + srcX3];

	//upbili_value =
	//	linear_X[0] * linear_Y[0] * downbili[srcY1 * dw + srcX1] +
	//	linear_X[0] * linear_Y[1] * downbili[srcY2 * dw + srcX1] +
	//	linear_X[1] * linear_Y[0] * downbili[srcY1 * dw + srcX2] +
	//	linear_X[1] * linear_Y[1] * downbili[srcY2 * dw + srcX2];

	upbicu_value = In[dstBaseY * ow + dstBaseX] + HF*(In[dstBaseY * ow + dstBaseX] - upbicu_value);

	if (dstBaseY < oh && dstBaseX < ow){
		Out[dstBaseY * ow + dstBaseX] = min(max(unsigned int(upbicu_value), 0), 255);
	}
}


void OptBicu_Bicuexeckernel(unsigned char * In, int iw, int ih,
	float * downbicu, int dw, int dh, float downscale_x, float downscale_y,
	unsigned char * Out, int ow, int oh, float upscale_x, float upscale_y
	){

	static dim3 grid;
	static dim3 block;

	block.x = 32;
	block.y = 32;
	block.z = 1;
	grid.x = (dw + block.x - 1) / block.x;
	grid.y = (dh + block.y - 1) / block.y;
	grid.z = 1;

	DownSample << <grid, block >> >(In, iw, ih, downbicu, dw, dh, downscale_x, downscale_y);

	grid.x = (ow + block.x - 1) / block.x;
	grid.y = (oh + block.y - 1) / block.y;
	grid.z = 1;


	//Sheme Four:
	//left side ==>  I + 4 *(I - upbicu(downbicu(I)))
	//right side ==> I
	UpEnhance << <grid, block >> >(downbicu, dw, dh, In, Out, ow, oh, upscale_x, upscale_y);
	printf("Here I anm\n\year");

}
